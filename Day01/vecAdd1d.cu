#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAStream.h>  // Add this header

__global__ void vecAddKernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int n){
  int i = threadIdx.x+ blockDim.x* blockIdx.x;
  if (i< n){
    C[i] = A[i] + B[i];
  }
}
__host__ torch::Tensor vecAdd(torch::Tensor tensor1, torch::Tensor tensor2){
  TORCH_CHECK(tensor1.is_cuda(), "Tensor must be a cuda tensor");
  TORCH_CHECK(tensor1.is_contiguous(), "Tensor must be contiguous");
  TORCH_CHECK(tensor2.is_cuda(), "Tensor must be a cuda tensor");
  TORCH_CHECK(tensor2.is_contiguous(), "Tensor must be contiguous");

  int size = tensor1.numel();
  float *data1 = tensor1.data_ptr<float>();
  float *data2 = tensor2.data_ptr<float>();
  torch::Tensor result = torch::empty_like(tensor1);
  float *data3 = result.data_ptr<float>();



  int threadsPerBlock = 256;
  int blocksPergrid = (size + threadsPerBlock -1/threadsPerBlock);
  hipStream_t stream = c10::cuda::getCurrentCUDAStream();
  vecAddKernel<<<blocksPergrid,threadsPerBlock,0,stream>>>(data1,data2,data3,size);
  C10_CUDA_CHECK(hipGetLastError());
  return result;


}
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("vec_add", &vecAdd, "Add two tensors");
}